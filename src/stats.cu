#include "hip/hip_runtime.h"
#include "cds/data_stats.h"

#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <vector>

using namespace std;

__global__ void calculate_totals(const float* data, const size_t record_count, const size_t field_count, float* totals) {
    extern __shared__ float sdata[];

    const unsigned int global_idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int global_idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int local_idx = threadIdx.x;

    sdata[threadIdx.x] = 0.0f;
    __syncthreads();

    float val = FP_NAN;
    if (global_idx_x < record_count && global_idx_y < field_count) {
        val = data[global_idx_x * field_count + global_idx_y];
        sdata[local_idx] = val;
    } else {
        return;
    }
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (local_idx < s) {
            sdata[local_idx] += sdata[local_idx + s];
        }
        __syncthreads();
    }

    if (local_idx == 0) {
        atomicAdd(&totals[global_idx_y], sdata[0]);
    }
}

__global__ void calculate_means_stddevs(const float* data, const size_t record_count, const size_t field_count,
    const float* totals, float* means, float* std_devs) {
    extern __shared__ float sdata[];

    const unsigned int global_idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int global_idx_y = blockIdx.y;
    const unsigned int local_idx = threadIdx.x;

    sdata[local_idx] = 0.0f;
    __syncthreads();

    float val = 0.0f;
    if (global_idx_x < record_count && global_idx_y < field_count) {
        val = data[global_idx_x * field_count + global_idx_y];
    }
    __syncthreads();

    if (local_idx == 0) {
        means[global_idx_y] = totals[global_idx_y] / static_cast<float>(record_count);
    }
    __syncthreads();

    if (global_idx_x < record_count && global_idx_y < field_count) {
        const float diff = val - means[global_idx_y];
        sdata[local_idx] = diff * diff;
    }
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (local_idx < s) {
            sdata[local_idx] += sdata[local_idx + s];
        }
        __syncthreads();
    }

    if (local_idx == 0) {
        atomicAdd(&std_devs[global_idx_y], sdata[0]);
    }
}

static void cleanup(float* data, float* totals, float* means, float* std_devs) {
    if (data != nullptr)
        hipFree(data);

    if (totals != nullptr)
        hipFree(totals);

    if (means != nullptr)
        hipFree(means);

    if (std_devs != nullptr)
        hipFree(std_devs);
}

// TODO: cleanup
bool calculate_stats(const vector<char>& data, const size_t field_count, const size_t record_count, DataStats& stats) {
    float *d_data, *d_totals, *d_means, *d_std_devs;

    if (hipMalloc(&d_data, data.size()) != hipSuccess) {
        cerr << "Error: hipMalloc failed for d_data" << endl;
        return false;
    }
    if (hipMemcpy(d_data, data.data(), data.size(), hipMemcpyHostToDevice) != hipSuccess) {
        cerr << "Error: hipMemcpy failed for d_data" << endl;
        cleanup(d_data, d_totals, d_means, d_std_devs);
        return false;
    }
    if (hipMalloc(&d_totals, field_count * sizeof(float)) != hipSuccess) {
        cerr << "Error: hipMalloc failed for d_totals" << endl;
        cleanup(d_data, d_totals, d_means, d_std_devs);
        return false;
    }
    if (hipMalloc(&d_means, field_count * sizeof(float)) != hipSuccess) {
        cerr << "Error: hipMalloc failed for d_means" << endl;
        cleanup(d_data, d_totals, d_means, d_std_devs);
        return false;
    }
    if (hipMalloc(&d_std_devs, field_count * sizeof(float)) != hipSuccess) {
        cerr << "Error: hipMalloc failed for d_std_devs" << endl;
        cleanup(d_data, d_totals, d_means, d_std_devs);
        return false;
    }

    hipError_t err = hipMemset(d_totals, 0, field_count * sizeof(float));
    if (err != hipSuccess) {
        cerr << "Error: hipMemset failed for d_totals: " << hipGetErrorString(err) << endl;
        cleanup(d_data, d_totals, d_means, d_std_devs);
        return false;
    }
    err = hipMemset(d_means, 0, field_count * sizeof(float));
    if (err != hipSuccess) {
        cerr << "Error: hipMemset failed for d_means: " << hipGetErrorString(err) << endl;
        return false;
    }
    err = hipMemset(d_std_devs, 0, field_count * sizeof(float));
    if (err != hipSuccess) {
        cerr << "Error: hipMemset failed for d_std_devs: " << hipGetErrorString(err) << endl;
        return false;
    }

    // TODO: handle case where record count is greater than max threads (implement batching)
    constexpr size_t block_width = 512; // TODO: make configurable
    const size_t block_count = (record_count + block_width - 1) / block_width;
    dim3 grid(block_count, field_count);
    dim3 block(block_width, 1);
    size_t shared_mem_size = block_width * sizeof(float);

    calculate_totals<<<grid, block, shared_mem_size>>>(d_data, record_count, field_count, d_totals);
    err = hipGetLastError();
    if (err != hipSuccess) {
        cerr << "Error: calculate_totals failed: " << hipGetErrorString(err) << endl;
        cleanup(d_data, d_totals, d_means, d_std_devs);
        return false;
    }

    if (hipDeviceSynchronize() != hipSuccess) {
        err = hipGetLastError();
        cerr << "Error: hipDeviceSynchronize failed: " << hipGetErrorString(err) << endl;
        cleanup(d_data, d_totals, d_means, d_std_devs);
        return false;
    }

    calculate_means_stddevs<<<grid, block, shared_mem_size>>>(d_data, record_count, field_count, d_totals, d_means, d_std_devs);
    err = hipGetLastError();
    if (err != hipSuccess) {
        cerr << "Error: calculate_means_stddevs failed: " << hipGetErrorString(err) << endl;
        cleanup(d_data, d_totals, d_means, d_std_devs);
        return false;
    }

    if (hipDeviceSynchronize() != hipSuccess) {
        err = hipGetLastError();
        cerr << "Error: hipDeviceSynchronize failed: " << hipGetErrorString(err) << endl;
        cleanup(d_data, d_totals, d_means, d_std_devs);
        return false;
    }

    float h_totals[field_count];
    if (hipMemcpy(h_totals, d_totals, field_count * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
        cerr << "Error: hipMemcpy failed for h_totals" << endl;
        cleanup(d_data, d_totals, d_means, d_std_devs);
        return false;
    }

    float h_means[field_count];
    if (hipMemcpy(h_means, d_means, field_count * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
        cerr << "Error: hipMemcpy failed for h_means" << endl;
        cleanup(d_data, d_totals, d_means, d_std_devs);
        return false;
    }

    float h_std_devs[field_count];
    if (hipMemcpy(h_std_devs, d_std_devs, field_count * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
        cerr << "Error: hipMemcpy failed for h_std_devs" << endl;
        cleanup(d_data, d_totals, d_means, d_std_devs);
        return false;
    }

    for (size_t i = 0; i < field_count; i++)
        h_std_devs[i] = sqrt(h_std_devs[i] / static_cast<float>(record_count));

    cleanup(d_data, d_totals, d_means, d_std_devs);

    stats.totals.insert(stats.totals.end(), h_totals, h_totals + field_count);
    stats.means.insert(stats.means.end(), h_means, h_means + field_count);
    stats.std_devs.insert(stats.std_devs.end(), h_std_devs, h_std_devs + field_count);

    return true;
}

