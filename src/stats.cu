#include "hip/hip_runtime.h"
/*******************************************************************************
 INCLUDES
*******************************************************************************/

#include "cds/stats.cuh"
#include "cds/math.cuh"
#include "cds/debug.h"

#include <cfloat>
#include <cstdarg>

/*******************************************************************************
 USINGS
*******************************************************************************/

using std::sqrt;

/*******************************************************************************
 KERNELS
*******************************************************************************/

__global__ void calculateMins(const float* data, const size_t recordCount, const size_t fieldCount,
    float* mins) {
    extern __shared__ float sdataMins[];

    const unsigned int globalIdxX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int globalIdxY = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int localIdx = threadIdx.x;

    sdataMins[localIdx] = FLT_MAX;
    __syncthreads();

    if (globalIdxX < recordCount && globalIdxY < fieldCount) {
        sdataMins[localIdx] = data[globalIdxX * fieldCount + globalIdxY];
    } else {
        return;
    }
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (localIdx < s) {
            sdataMins[localIdx] = fminf(sdataMins[localIdx], sdataMins[localIdx + s]);
        }
        __syncthreads();
    }

    if (localIdx == 0) {
        atomicMinFloat(&mins[globalIdxY], sdataMins[0]);
    }
}

__global__ void calculateMaxs(const float* data, const size_t recordCount, const size_t fieldCount,
    float* maxs) {
    extern __shared__ float sdataMaxs[];

    const unsigned int globalIdxX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int globalIdxY = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int localIdx = threadIdx.x;

    sdataMaxs[localIdx] = FLT_MIN;
    __syncthreads();

    if (globalIdxX < recordCount && globalIdxY < fieldCount) {
        sdataMaxs[localIdx] = data[globalIdxX * fieldCount + globalIdxY];
    } else {
        return;
    }
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (localIdx < s) {
            sdataMaxs[localIdx] = fmaxf(sdataMaxs[localIdx], sdataMaxs[localIdx + s]);
        }
        __syncthreads();
    }

    if (localIdx == 0) {
        atomicMaxFloat(&maxs[globalIdxY], sdataMaxs[0]);
    }
}

__global__ void calculateTotals(const float* data, const size_t recordCount, const size_t fieldCount,
    float* totals) {
    extern __shared__ float sdataTotals[];

    const unsigned int globalIdxX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int globalIdxY = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int localIdx = threadIdx.x;

    sdataTotals[localIdx] = 0.0f;
    __syncthreads();

    if (globalIdxX < recordCount && globalIdxY < fieldCount) {
        sdataTotals[localIdx] = data[globalIdxX * fieldCount + globalIdxY];
    } else {
        return;
    }
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (localIdx < s) {
            sdataTotals[localIdx] += sdataTotals[localIdx + s];
        }
        __syncthreads();
    }

    if (localIdx == 0) {
        atomicAdd(&totals[globalIdxY], sdataTotals[0]);
    }
}

__global__ void convertDataToDeltas(float* data, const size_t recordCount, const size_t fieldCount) {
    const unsigned int globalIdxX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int globalIdxY = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int globalIdx = globalIdxX * fieldCount + globalIdxY;

    float localValue;
    float otherValue;
    if (globalIdxX < recordCount && globalIdxY < fieldCount) {
        localValue = data[globalIdx];
        if (globalIdxX == 0)
            otherValue = data[(globalIdxX + 1) * fieldCount + globalIdxY];
        else
            otherValue = data[(globalIdxX - 1) * fieldCount + globalIdxY];
    } else {
        return;
    }
    __syncthreads();

    data[globalIdx] = abs(otherValue - localValue);
}

__global__ void convertDeltasToDeltasSquared(float* data, const size_t recordCount, const size_t fieldCount) {
    const unsigned int globalIdxX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int globalIdxY = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int globalIdx = globalIdxX * fieldCount + globalIdxY;

    if (globalIdxX < recordCount && globalIdxY < fieldCount) {
        data[globalIdx] = pow(data[globalIdx], 2);
    }
}

__global__ void calculateMeansStddevs(const float* data, const size_t recordCount, const size_t fieldCount,
    const float* totals, float* means, float* stdDevs) {
    extern __shared__ float sdataStddev[];

    const unsigned int globalIdxX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int globalIdxY = blockIdx.y;
    const unsigned int localIdx = threadIdx.x;

    sdataStddev[localIdx] = 0.0f;
    __syncthreads();

    float val = 0.0f;
    if (globalIdxX < recordCount && globalIdxY < fieldCount) {
        val = data[globalIdxX * fieldCount + globalIdxY];
    }
    __syncthreads();

    if (localIdx == 0) {
        means[globalIdxY] = totals[globalIdxY] / static_cast<float>(recordCount);
    }
    __syncthreads();

    if (globalIdxX < recordCount && globalIdxY < fieldCount) {
        const float diff = val - means[globalIdxY];
        sdataStddev[localIdx] = diff * diff;
    }
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (localIdx < s) {
            sdataStddev[localIdx] += sdataStddev[localIdx + s];
        }
        __syncthreads();
    }

    if (localIdx == 0) {
        atomicAdd(&stdDevs[globalIdxY], sdataStddev[0]);
    }
}

/*******************************************************************************
 KERNEL WRAPPERS
*******************************************************************************/

namespace {
    bool runCalculateMins(dim3 grid, dim3 block, size_t sharedMemorySize, hipStream_t stream, const float* data,
        const size_t recordCount, const size_t fieldCount, float* mins) {
            calculateMins<<<grid, block, sharedMemorySize, stream>>>(data, recordCount, fieldCount, mins);
            if (const hipError_t err = hipGetLastError(); err != hipSuccess) {
                ERROR("calculateMins failed: %s", hipGetErrorString(err));
                return false;
            }
            return true;
    }

    bool runCalculateMaxs(dim3 grid, dim3 block, size_t sharedMemorySize, hipStream_t stream, const float* data,
        const size_t recordCount, const size_t fieldCount, float* maxs) {
            calculateMaxs<<<grid, block, sharedMemorySize, stream>>>(data, recordCount, fieldCount, maxs);
            if (const hipError_t err = hipGetLastError(); err != hipSuccess) {
                ERROR("calculateMaxs failed: %s", hipGetErrorString(err));
                return false;
            }
            return true;
    }

    bool runCalculateTotals(dim3 grid, dim3 block, size_t sharedMemorySize, hipStream_t stream, const float* data,
        const size_t recordCount, const size_t fieldCount, float* totals) {
            calculateTotals<<<grid, block, sharedMemorySize, stream>>>(data, recordCount, fieldCount, totals);
            if (const hipError_t err = hipGetLastError(); err != hipSuccess) {
                ERROR("calculateTotals failed: %s", hipGetErrorString(err));
                return false;
            }
            return true;
    }

    bool runCalculateMeansStddevs(dim3 grid, dim3 block, size_t sharedMemorySize, hipStream_t stream,
        const float* data, const size_t recordCount, const size_t fieldCount, const float* totals,
        float* means, float* stdDevs) {
            calculateMeansStddevs<<<grid, block, sharedMemorySize, stream>>>(data, recordCount, fieldCount, totals, means, stdDevs);
            if (const hipError_t err = hipGetLastError(); err != hipSuccess) {
                ERROR("calculateMeansStddevs failed: %s", hipGetErrorString(err));
                return false;
            }
            return true;
    }

    bool runConvertDataToDeltas(dim3 grid, dim3 block, size_t sharedMemorySize, hipStream_t stream,
        float *data, const size_t recordCount, const size_t fieldCount) {
        convertDataToDeltas<<<grid, block, sharedMemorySize, stream>>>(data, recordCount, fieldCount);
        if (const hipError_t err = hipGetLastError(); err != hipSuccess) {
            ERROR("convertDataToDeltas failed: %s", hipGetErrorString(err));
            return false;
        }
        return true;
    }
}

/*******************************************************************************
 UTILITY FUNCTIONS
*******************************************************************************/

namespace {
    bool initializeMemory(float* dPtr, const int value, const size_t size, const char* name) {
        if (const hipError_t err = hipMemset(dPtr, value, size); err != hipSuccess) {
            ERROR("hipMemset failed for %s: %s", name, hipGetErrorString(err));
            return false;
        }
        return true;
    }

    bool freeMemory(const int count, ...) {
        va_list args;
        va_start(args, count);

        for (int i = 0; i < count; i++) {
            if (float* deviceArray = va_arg(args, float*); deviceArray != nullptr) {
                if (const hipError_t err = hipFree(deviceArray); err != hipSuccess) {
                    ERROR("hipFree failed: %s", hipGetErrorString(err));
                    return false;
                }
            }
        }

        va_end(args);
        return true;
    }

    bool copyToDevice(float* dPtr, const char* data, const size_t dataSize) {
        if (const hipError_t err = hipMemcpy(dPtr, data, dataSize, hipMemcpyHostToDevice); err != hipSuccess) {
            ERROR("copyToDevice failed for data: %s", hipGetErrorString(err));
            return false;
        }
        return true;
    }

    bool copyFromDevice(float* hPtr, const float* dPtr, const size_t size, const char* name) {
        if (const hipError_t err = hipMemcpy(hPtr, dPtr, size, hipMemcpyDeviceToHost); err != hipSuccess) {
            ERROR("copyFromDevice failed for %s: %s", name, hipGetErrorString(err));
            return false;
        }
        return true;
    }

    bool allocateMemory(float*& dPtr, const size_t size, const char* name) {
        if (const hipError_t err = hipMalloc(&dPtr, size); err != hipSuccess) {
            ERROR("hipMalloc failed for %s: %s", name, hipGetErrorString(err));
            return false;
        }
        return true;
    }

    bool allocateMemory(const char* data, const size_t dataSize, const DataStats& stats, float*& dData,
        float*& dMins, float*& dMaxs, float*& dTotals, float*& dMeans, float*& dStdDevs,
        float*& dDeltaMins, float*& dDeltaMaxs, float*& dDeltaTotals, float*& dDeltaMeans, float*& dDeltaStdDevs) {
        const auto statsSize = stats.fieldCount * sizeof(float);

        if (!allocateMemory(dData, dataSize, "dData")) {
            return false;
        }
        if (!copyToDevice(dData, data, dataSize)) {
            freeMemory(1, dData);
            return false;
        }
        if (!allocateMemory(dMins, statsSize, "dMins")) {
            freeMemory(1, dData);
            return false;
        }
        if (!allocateMemory(dMaxs, statsSize, "dMaxs")) {
            freeMemory(2, dData, dMins);
            return false;
        }
        if (!allocateMemory(dTotals, statsSize, "dTotals")) {
            freeMemory(3, dData, dMins, dMaxs);
            return false;
        }
        if (!allocateMemory(dMeans, statsSize, "dMeans")) {
            freeMemory(4, dData, dMins, dMaxs, dTotals);
            return false;
        }
        if (!allocateMemory(dStdDevs, statsSize, "dStdDevs")) {
            freeMemory(5, dData, dMins, dMaxs, dTotals, dMeans);
            return false;
        }
        if (!allocateMemory(dDeltaMins, statsSize, "dDeltaMins")) {
            freeMemory(6, dData, dMins, dMaxs, dTotals, dMeans, dStdDevs);
            return false;
        }
        if (!allocateMemory(dDeltaMaxs, statsSize, "dDeltaMaxs")) {
            freeMemory(7, dData, dMins, dMaxs, dTotals, dMeans, dStdDevs,
                dDeltaMins);
            return false;
        }
        if (!allocateMemory(dDeltaTotals, statsSize, "dDeltaTotals")) {
            freeMemory(8, dData, dMins, dMaxs, dTotals, dMeans, dStdDevs,
                dDeltaMins, dDeltaMaxs);
            return false;
        }
        if (!allocateMemory(dDeltaMeans, statsSize, "dDeltaMeans")) {
            freeMemory(9, dData, dMins, dMaxs, dTotals, dMeans, dStdDevs,
                dDeltaMins, dDeltaMaxs, dDeltaTotals);
            return false;
        }
        if (!allocateMemory(dDeltaStdDevs, statsSize, "dDeltaStdDevs")) {
            freeMemory(10, dData, dMins, dMaxs, dTotals, dMeans, dStdDevs,
                dDeltaMins, dDeltaMaxs, dDeltaTotals, dDeltaMeans);
            return false;
        }

        return true;
    }

    bool synchronizeDevice() {
        if (const hipError_t err = hipDeviceSynchronize(); err != hipSuccess) {
            ERROR("hipDeviceSynchronize failed: %s", hipGetErrorString(err));
            return false;
        }
        return true;
    }

    void finishCalculatingStdDevs(const size_t fieldCount, const size_t recordCount, float* stdDevs) {
        for (size_t i = 0; i < fieldCount; i++) {
            stdDevs[i] = sqrt(stdDevs[i] / static_cast<float>(recordCount));
        }
    }
}

/*******************************************************************************
 INTERNAL FUNCTIONS
*******************************************************************************/

bool calculateStats(const char *data, const size_t dataSize, DataStats &stats) {
    float *dData;
    float *dMins, *dMaxs, *dTotals, *dMeans, *dStdDevs;
    float *dDeltaMins, *dDeltaMaxs, *dDeltaTotals, *dDeltaMeans, *dDeltaStdDevs;

    if (!allocateMemory(data, dataSize, stats, dData, dMins, dMaxs, dTotals, dMeans, dStdDevs,
        dDeltaMins, dDeltaMaxs, dDeltaTotals, dDeltaMeans, dDeltaStdDevs)) {
        return false;
    }

    const auto statsSize = stats.fieldCount * sizeof(float);
    if (!initializeMemory(dMins, INT_MAX, statsSize, "dMins") ||
        !initializeMemory(dMaxs, INT_MIN, statsSize, "dMaxs") ||
        !initializeMemory(dTotals, 0, statsSize, "dTotals") ||
        !initializeMemory(dMeans, 0, statsSize, "dMeans") ||
        !initializeMemory(dStdDevs, 0, statsSize, "dStdDevs") ||
        !initializeMemory(dDeltaMins, INT_MAX, statsSize, "dDeltaMins") ||
        !initializeMemory(dDeltaMaxs, INT_MIN, statsSize, "dDeltaMaxs") ||
        !initializeMemory(dDeltaTotals, 0, statsSize, "dDeltaTotals") ||
        !initializeMemory(dDeltaMeans, 0, statsSize, "dDeltaMeans") ||
        !initializeMemory(dDeltaStdDevs, 0, statsSize, "dDeltaStdDevs")) {
        freeMemory(11, dData, dMins, dMaxs, dTotals, dMeans, dStdDevs,
            dDeltaMins, dDeltaMaxs, dDeltaTotals, dDeltaMeans, dDeltaStdDevs);
        return false;
    }

    // TODO: handle case where record count is greater than max threads (implement batching)
    constexpr size_t blockWidth = 32; // TODO: make configurable
    const size_t blockCount = (stats.recordCount + blockWidth - 1) / blockWidth;
    const dim3 grid(blockCount, stats.fieldCount);
    constexpr dim3 block(blockWidth, 1);
    constexpr size_t sharedMemSize = blockWidth * sizeof(float);

    hipStream_t minStream, maxStream;
    hipStreamCreate(&minStream);
    hipStreamCreate(&maxStream);

    if (!runCalculateMins(grid, block, sharedMemSize, minStream, dData, stats.recordCount, stats.fieldCount, dMins) ||
        !runCalculateMaxs(grid, block, sharedMemSize, maxStream, dData, stats.recordCount, stats.fieldCount, dMaxs) ||
        !runCalculateTotals(grid, block, sharedMemSize, nullptr, dData, stats.recordCount, stats.fieldCount, dTotals) ||
        !synchronizeDevice() ||
        !runCalculateMeansStddevs(grid, block, sharedMemSize, nullptr, dData, stats.recordCount, stats.fieldCount, dTotals, dMeans, dStdDevs) ||
        !synchronizeDevice()) {
        freeMemory(11, dData, dMins, dMaxs, dTotals, dMeans, dStdDevs,
            dDeltaMins, dDeltaMaxs, dDeltaTotals, dDeltaMeans, dDeltaStdDevs);
        return false;
    }

    if (!runConvertDataToDeltas(grid, block, sharedMemSize, nullptr, dData, stats.recordCount, stats.fieldCount) ||
        !synchronizeDevice()) {
        freeMemory(11, dData, dMins, dMaxs, dTotals, dMeans, dStdDevs,
            dDeltaMins, dDeltaMaxs, dDeltaTotals, dDeltaMeans, dDeltaStdDevs);
        return false;
    }

    if (!runCalculateMins(grid, block, sharedMemSize, minStream, dData, stats.recordCount, stats.fieldCount, dDeltaMins) ||
        !runCalculateMaxs(grid, block, sharedMemSize, maxStream, dData, stats.recordCount, stats.fieldCount, dDeltaMaxs) ||
        !runCalculateTotals(grid, block, sharedMemSize, nullptr, dData, stats.recordCount, stats.fieldCount, dDeltaTotals) ||
        !synchronizeDevice() ||
        !runCalculateMeansStddevs(grid, block, sharedMemSize, nullptr, dData, stats.recordCount, stats.fieldCount, dDeltaTotals, dDeltaMeans, dDeltaStdDevs) ||
        !synchronizeDevice()) {
        freeMemory(11, dData, dMins, dMaxs, dTotals, dMeans, dStdDevs,
            dDeltaMins, dDeltaMaxs, dDeltaTotals, dDeltaMeans, dDeltaStdDevs);
        return false;
    }

    float hMins[stats.fieldCount], hMaxs[stats.fieldCount],
        hTotals[stats.fieldCount], hMeans[stats.fieldCount], hStdDevs[stats.fieldCount];

    float hDeltaMins[stats.fieldCount], hDeltaMaxs[stats.fieldCount],
            hDeltaTotals[stats.fieldCount], hDeltaMeans[stats.fieldCount], hDeltaStdDevs[stats.fieldCount];

    if (!copyFromDevice(hMins, dMins, statsSize, "hMins") ||
        !copyFromDevice(hMaxs, dMaxs, statsSize, "hMaxs") ||
        !copyFromDevice(hTotals, dTotals, statsSize, "hTotals") ||
        !copyFromDevice(hMeans, dMeans, statsSize, "hMeans") ||
        !copyFromDevice(hStdDevs, dStdDevs, statsSize, "hStdDevs")) {
        freeMemory(11, dData, dMins, dMaxs, dTotals, dMeans, dStdDevs,
            dDeltaMins, dDeltaMaxs, dDeltaTotals, dDeltaMeans, dDeltaStdDevs);
        return false;
    }

    if (!copyFromDevice(hDeltaMins, dDeltaMins, statsSize, "hDeltaMins") ||
        !copyFromDevice(hDeltaMaxs, dDeltaMaxs, statsSize, "hDeltaMaxs") ||
        !copyFromDevice(hDeltaTotals, dDeltaTotals, statsSize, "hDeltaTotals") ||
        !copyFromDevice(hDeltaMeans, dDeltaMeans, statsSize, "hDeltaMeans") ||
        !copyFromDevice(hDeltaStdDevs, dDeltaStdDevs, statsSize, "hDeltaStdDevs")) {
        freeMemory(11, dData, dMins, dMaxs, dTotals, dMeans, dStdDevs,
            dDeltaMins, dDeltaMaxs, dDeltaTotals, dDeltaMeans, dDeltaStdDevs);
        return false;
    }

    finishCalculatingStdDevs(stats.fieldCount, stats.recordCount, hStdDevs);
    finishCalculatingStdDevs(stats.fieldCount, stats.recordCount, hDeltaStdDevs);

    freeMemory(11, dData, dMins, dMaxs, dTotals, dMeans, dStdDevs,
        dDeltaMins, dDeltaMaxs, dDeltaTotals, dDeltaMeans, dDeltaStdDevs);

    stats.minimums.insert(stats.minimums.end(), hMins, hMins + stats.fieldCount);
    stats.maximums.insert(stats.maximums.end(), hMaxs, hMaxs + stats.fieldCount);
    stats.totals.insert(stats.totals.end(), hTotals, hTotals + stats.fieldCount);
    stats.means.insert(stats.means.end(), hMeans, hMeans + stats.fieldCount);
    stats.stdDevs.insert(stats.stdDevs.end(), hStdDevs, hStdDevs + stats.fieldCount);

    stats.deltaMinimums.insert(stats.deltaMinimums.end(), hDeltaMins, hDeltaMins + stats.fieldCount);
    stats.deltaMaximums.insert(stats.deltaMaximums.end(), hDeltaMaxs, hDeltaMaxs + stats.fieldCount);
    stats.deltaTotals.insert(stats.deltaTotals.end(), hDeltaTotals, hDeltaTotals + stats.fieldCount);
    stats.deltaMeans.insert(stats.deltaMeans.end(), hDeltaMeans, hDeltaMeans + stats.fieldCount);
    stats.deltaStdDevs.insert(stats.deltaStdDevs.end(), hDeltaStdDevs, hDeltaStdDevs + stats.fieldCount);

    return true;
}
